#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.h"

// host implementation of dot product
double dot_host(const double *x, const double* y, int n) {
    double sum = 0;
    for(auto i=0; i<n; ++i) {
        sum += x[i]*y[i];
    }
    return sum;
}

// for handling non-powers-of-two we need some helpers
// for detecting power of two and rounding up to the next
// power of two
__device__
bool is_power_of_two(int n) {
    return (n & (n - 1)) == 0;
}

__device__
int next_power_of_two(int n) {
    if (n < 0) {
        return 0;
    }
    --n;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    return n+1;
}

// solution for arbitrary n<=1024
__global__
void dot_gpu_kernel(const double *x, const double* y, double *result, int n) {
    extern __shared__ double buffer[];

    int idx = threadIdx.x;

    // all threads set buffer to zero, regardless of whether
    // they will participate in the reduction
    // this ensures that the binary reduction step works
    buffer[idx] = 0.;
    __syncthreads();

    if(idx<n) {
        buffer[idx] = x[idx]*y[idx];
        __syncthreads();

        if(!is_power_of_two(n)) {
            n = next_power_of_two(n);
        }
        auto m = n/2;
        while(m) {
            __syncthreads();
            if(idx<m) {
                buffer[idx] += buffer[idx+m];
            }

            m /= 2;
        }

        if(idx==0) {
            result[0] = buffer[0];
        }
    }
}

double dot_gpu(const double *x, const double* y, int n) {
    static double* result = malloc_device<double>(1);
    double r;
    dot_gpu_kernel<<<1, 1024, sizeof(double)*1024>>>(x, y, result, n);
    copy_to_host<double>(result, &r, 1);
    return r;
}

int main(int argc, char** argv) {
    size_t n  = read_arg(argc, argv, 1, 4);

    auto size_in_bytes = n * sizeof(double);

    std::cout << "dot product CUDA of length n = " << n
              << " : " << size_in_bytes/(1024.*1024.) << "MB"
              << std::endl;

    hipInit(0);

    auto x_h = malloc_host<double>(n, 2.);
    auto y_h = malloc_host<double>(n);
    for(auto i=0; i<n; ++i) {
        y_h[i] = rand()%10;
    }

    auto x_d = malloc_device<double>(n);
    auto y_d = malloc_device<double>(n);

    // copy initial conditions to device
    copy_to_device<double>(x_h, x_d, n);
    copy_to_device<double>(y_h, y_d, n);

    auto result   = dot_gpu(x_d, y_d, n);
    auto expected = dot_host(x_h, y_h, n);
    std::cout << "expected " << expected << " got " << result << std::endl;

    return 0;
}

